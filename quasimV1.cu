#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

float *fetch_data(float (&u)[2][2], char* filename, int &qno, int &vec_length)
{
    FILE *file;
    bool bfirst = true;
    file = fopen(filename, "r");
    int i, j;
    char c;
    float rvar;
    float *local;
    float *vec;
    if (file == NULL)
    {
        printf("File Not Found!");
        return vec;
    }
    
    // Read the U matrix
    fscanf(file, "%f %f", &u[0][0], &u[0][1]);
    fscanf(file, "%f %f", &u[1][0], &u[1][1]);

    // Read the vector 
    i = 0;
    while (fscanf(file, "%f", &rvar) == 1)
    {
        if (bfirst)
        {
            local = (float *) malloc (sizeof(float));
            local[i] = rvar;
            i++;
            bfirst = false;
        }
        else
        {
            local = (float *) realloc (local, (i+1)*sizeof(float));
            local[i] = rvar;
            i++;
        }
    }
    
    qno = (int) local[i-1];
    vec = (float *) malloc ((i-1) * sizeof(float));
    vec_length = (i - 1);    

    for (j = 0; j < i-1; j++)
    {
        vec[j] = local[j];

    }
    
    free(local);
    return vec;
}

__global__ void
quantum_gate(float (&u)[2][2], float* vec)
{
    int i;
}


int main(int argc, char *argv[])
{
    float u[2][2];
    hipError_t err = hipSuccess;
    
    int qno, vec_length;
    char* filename = argv[1];

    printf("%s\n", filename);
    
    float *vec = fetch_data(u, filename, qno, vec_length);

    for (int j = 0; j < vec_length; j++)
    {
        printf("%f\n", vec[j]);
    }
    
    printf("%f %f %f %f %d %d\n", u[0][0], u[0][1], u[1][0], u[1][1], qno, vec_length);
    free(vec);
    return 1;

}