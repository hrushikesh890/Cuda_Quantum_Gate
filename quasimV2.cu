#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>

float *fetch_data(float u[4], char* filename, int &qno, int &vec_length)
{
    FILE *file;
    bool bfirst = true;
    file = fopen(filename, "r");
    int i, j;
    char c;
    float rvar;
    float *local;
    float *vec;
    if (file == NULL)
    {
        printf("File Not Found!");
        return vec;
    }
    
    // Read the U matrix
    fscanf(file, "%f %f", &u[0], &u[1]);
    fscanf(file, "%f %f", &u[2], &u[3]);

    // Read the vector 
    i = 0;
    while (fscanf(file, "%f", &rvar) == 1)
    {
        if (bfirst)
        {
            local = (float *) malloc (sizeof(float));
            local[i] = rvar;
            i++;
            bfirst = false;
        }
        else
        {
            local = (float *) realloc (local, (i+1)*sizeof(float));
            local[i] = rvar;
            i++;
        }
    }
    
    qno = (int) local[i-1];
    vec = (float *) malloc ((i-1) * sizeof(float));
    vec_length = (i - 1);    

    for (j = 0; j < i-1; j++)
    {
        vec[j] = local[j];

    }
    
    free(local);
    return vec;
}

__global__ void
quantum_gate_multiply(float* u, float* vec, float* op, int qno, int vec_length, int pw, int mask, int antimask)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int idx1, idx2;

    idx1 = (i & mask) | ((i & antimask) << 1);
    idx2 = (idx1 | (1 << qno));

    op[idx1] = u[0] * vec[idx1] + u[1] * vec[idx2];
    op[idx2] = u[2] * vec[idx1] + u[3] * vec[idx2];
}


int main(int argc, char *argv[])
{
    hipError_t err = hipSuccess;
    float* u;
    //printf("HERE\n");
    err = hipMallocManaged((void **)&u, (4 * sizeof(float)));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device arr A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    int qno, vec_length;
    char* filename = argv[1];

    //printf("%s\n", filename);
    
    float *vec1 = fetch_data(u, filename, qno, vec_length);

    float *vec;
    err = hipMallocManaged(&vec, (vec_length * sizeof(float)));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allcoate vector vec (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    float *op;
    err = hipMallocManaged(&op, (vec_length * sizeof(float)));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allcoate vector op (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < vec_length; i++)
    {
        vec[i] = vec1[i];
        op[i] = 0;
    }

    int mask = 0;
    int pw = (int)(log(vec_length)/log(2));
    int antimask = (int)(pow(2, (pw - 1)) - 1);
    
    for (int i = 0; i < qno; i++)
    {
        mask = ((mask << 1) | 1);
    }

    antimask = (antimask & (~mask));

    int threadsPerBlock = 256;
    int blocksPerGrid =((vec_length/2) + threadsPerBlock - 1) / threadsPerBlock;
    
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    //vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    //quantum_gate_multiply(u, vec, op, qno, vec_length, pw, mask, antimask);
    
    struct timeval begin, end; 
    gettimeofday (&begin, NULL);

    quantum_gate_multiply<<<blocksPerGrid, threadsPerBlock>>>(u, vec, op, qno, vec_length, pw, mask, antimask);
    hipDeviceSynchronize();
    gettimeofday (&end, NULL);

    //printf("%d\n", vec_length);
    for (int j = 0; j < vec_length; j++)
    {
        printf("%.3f\n", op[j]);
    }

    int time_in_us = 1e6 * (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec);
    printf("Time - %d", time_in_us);

    free(vec1);
    //free(op);
    return 1;
}